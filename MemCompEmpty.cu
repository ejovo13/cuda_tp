/**************************************************************
This code compares standard CPU allocation with the locked one.
It also checks the effeciency of mapping the CPU memory 

This code is a part of a course on cuda taught by the author: 
Lokman A. Abbas-Turki

Those who re-use this code should mention in their code 
the name of the author above.
***************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

// This kernel is needed to compare the mapped memory to other memories
__global__ void test_kernel(int *Tab, int size, int i){

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x<size){
		Tab[x] = i;
	}
}

float malloc_trans(int size, int NbT, bool flag) {

	int *a, *aGPU;
	float TimeVar;
	hipEvent_t start, stop;
	testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));

	a = (int*)malloc(size*sizeof(int));
	testCUDA(hipMalloc(&aGPU,size*sizeof(int)));

	testCUDA(hipEventRecord(start,0));

	for (int i=0; i<NbT; i++) {
		if (flag){
			testCUDA(hipMemcpy(aGPU, a, size*sizeof(int),	hipMemcpyHostToDevice)); 
			test_kernel<<<(size+511)/512, 512>>>(aGPU,size,i);//Comparison with mapped
		}else{
			test_kernel<<<(size+511)/512, 512>>>(aGPU,size,i);//Comparison with mapped
			testCUDA(hipMemcpy(a, aGPU, size*sizeof(int),	hipMemcpyDeviceToHost));
		}
	}

	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	testCUDA(hipFree(aGPU));
	free(a);	
	return TimeVar;
}

float hostAlloc_trans(int size, int NbT, bool flag) {

	
}

float mappedAlloc_trans(int size, int NbT) {

	
}

float Unified_trans(int size, int NbT) {

	int* a;
	float TimeVar;
	hipEvent_t start, stop;
	testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));

	testCUDA(hipMallocManaged(&a, size*sizeof(float)));

	testCUDA(hipEventRecord(start, 0));

	for (int i = 0; i < NbT; i++) {
		test_kernel <<<(size + 511)/512, 512>>> (a, size, i);
	}

	testCUDA(hipDeviceSynchronize());

	testCUDA(hipEventRecord(stop, 0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	printf("Check value %i\n", a[211]);
	testCUDA(hipFree(a));
	return TimeVar;
}

int main (void){

	int size = 1024*1024*8;
	int NbT = 100;
	float TimeVar;

	testCUDA(hipSetDeviceFlags(hipDeviceMapHost));

	TimeVar = malloc_trans(size, NbT, true);
	printf("Processing time when using malloc CPU2GPU: %f s\n", 
		   0.001f*TimeVar);
	TimeVar = malloc_trans(size, NbT, false);
	printf("Processing time when using malloc GPU2CPU: %f s\n", 
		   0.001f*TimeVar);

	TimeVar = hostAlloc_trans(size, NbT, true);
	printf("Processing time when using hipHostAlloc CPU2GPU: %f s\n", 
		   0.001f*TimeVar);
	TimeVar = hostAlloc_trans(size, NbT, false);
	printf("Processing time when using hipHostAlloc GPU2CPU: %f s\n", 
		   0.001f*TimeVar);

	TimeVar = mappedAlloc_trans(size, NbT);
	printf("Processing time for mapped memory: %f s\n", 
		   0.001f*TimeVar);

	//TimeVar = Unified_trans(size, NbT);
	//printf("Processing time for unified memory: %f s\n",
	//	0.001f * TimeVar);

	return 0;
}

